#include "hip/hip_runtime.h"
//
// Experimental Kernel for Kepler (Compute 3.5) devices
//
// Eliminates shared memory entirely, uses warp shuffle instructions instead
// based on a technique found in this blog posting by Allan MacKinnon:
// http://www.pixel.io/blog/2013/4/7/fast-matrix-transposition-without-shuffling-or-shared-memory.html
//
// Does not yet run as fast as the shared memory based kernel, but there may
// be further room for optimization! (417 kHash/s vs 450 kHash/s for T kernel)
// The card also seems to run hotter, running into its thermal limits sooner.
//
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "test_kernel.h"

// grab lane ID
static __device__ __inline__ unsigned int __laneId() { unsigned int laneId; asm( "mov.u32 %0, %%laneid;" : "=r"( laneId ) ); return laneId; }

// forward references
__global__ void test_scrypt_core_kernelA(uint32_t *g_idata);
__global__ void test_scrypt_core_kernelB(uint32_t *g_odata);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

TestKernel::TestKernel() : KernelInterface()
{
}

void TestKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool TestKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    test_scrypt_core_kernelA<<< grid, threads, 0, stream >>>(d_idata);

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    test_scrypt_core_kernelB<<< grid, threads, 0, stream >>>(d_odata);

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    left.z ^= right.z;
    left.w ^= right.w;
    return left;
}

__device__ __forceinline__ uint4 __shfl(const uint4 val, unsigned int lane)
{
    return make_uint4(
        (unsigned int)__shfl((int)val.x, lane),
        (unsigned int)__shfl((int)val.y, lane),
        (unsigned int)__shfl((int)val.z, lane),
        (unsigned int)__shfl((int)val.w, lane));
}

__device__ __forceinline__ void __swap(uint4 &a, uint4 &b)
{
//    uint4 t = b; b = a; a = t;
    uint32_t t;
    t=a.x; a.x=b.x; b.x=t;
    t=a.y; a.y=b.y; b.y=t;
    t=a.z; a.z=b.z; b.z=t;
    t=a.w; a.w=b.w; b.w=t;
}

__device__ __forceinline__ void __transposed_write(uint4 (&S)[4], uint4 *D, int spacing=1)
{
    unsigned int laneId = __laneId();

    unsigned int lane4 = laneId%4;
    unsigned int tile  = laneId/4;
    unsigned int tile4 = tile*4;

    unsigned int rot3 = tile4+(lane4+3)%4;
    unsigned int rot2 = tile4+(lane4+2)%4;
    unsigned int rot1 = tile4+(lane4+1)%4;

    // rotate
    S[1] = __shfl(S[1], rot3);
    S[2] = __shfl(S[2], rot2);
    S[3] = __shfl(S[3], rot1);

    // exchange
    if (lane4 >= 2) { __swap(S[0], S[2]); __swap(S[1], S[3]); }

    // select + write
    D[spacing*2*(16*tile   )+ lane4     ] = (laneId % 2 == 0) ? S[0] : S[1];
    D[spacing*2*(16*tile+4 )+(lane4+3)%4] = (laneId % 2 == 0) ? S[3] : S[0];
    D[spacing*2*(16*tile+8 )+(lane4+2)%4] = (laneId % 2 == 0) ? S[2] : S[3];
    D[spacing*2*(16*tile+12)+(lane4+1)%4] = (laneId % 2 == 0) ? S[1] : S[2];

    // undo exchange
    if (lane4 >= 2) { __swap(S[0], S[2]); __swap(S[1], S[3]); }

    // undo rotate
    S[1] = __shfl(S[1], rot1);
    S[2] = __shfl(S[2], rot2);
    S[3] = __shfl(S[3], rot3);
}

__device__ __forceinline__ void __transposed_read(uint4 *S, uint4 (&D)[4], int spacing=1)
{
    unsigned int laneId = __laneId();

    unsigned int lane4 = laneId%4;
    unsigned int tile  = laneId/4;
    unsigned int tile4 = tile*4;

    unsigned int rot3 = tile4+(lane4+3)%4;
    unsigned int rot2 = tile4+(lane4+2)%4;
    unsigned int rot1 = tile4+(lane4+1)%4;

    // read and select
    uint4 tmp; 
    tmp = __ldg(&S[spacing*2*(16*tile   )+ lane4     ]); if (laneId % 2 == 0) D[0] = tmp; else D[1] = tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+4 )+(lane4+3)%4]); if (laneId % 2 == 0) D[3] = tmp; else D[0] = tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+8 )+(lane4+2)%4]); if (laneId % 2 == 0) D[2] = tmp; else D[3] = tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+12)+(lane4+1)%4]); if (laneId % 2 == 0) D[1] = tmp; else D[2] = tmp;

    // undo exchange
    if (lane4 >= 2) { __swap(D[0], D[2]); __swap(D[1], D[3]); }

    // undo rotate
    D[1] = __shfl(D[1], rot1);
    D[2] = __shfl(D[2], rot2);
    D[3] = __shfl(D[3], rot3);
}

__device__ __forceinline__ void __transposed_xor(uint4 *S, uint4 (&D)[4], int spacing=1, int row=0)
{
    unsigned int laneId = __laneId();

    unsigned int lane4 = laneId%4;
    unsigned int tile  = laneId/4;
    unsigned int tile4 = tile*4;

    unsigned int rot3 = tile4+(lane4+3)%4;
    unsigned int rot2 = tile4+(lane4+2)%4;
    unsigned int rot1 = tile4+(lane4+1)%4;

    // rotate
    D[1] = __shfl(D[1], rot3);
    D[2] = __shfl(D[2], rot2);
    D[3] = __shfl(D[3], rot1);

    // exchange
    if (lane4 >= 2) { __swap(D[0], D[2]); __swap(D[1], D[3]); }

    // read and select
    uint4 tmp; 
    tmp = __ldg(&S[spacing*2*(16*tile   )+ lane4     +8*__shfl(row,tile4  )]); if (laneId % 2 == 0) D[0] ^= tmp; else D[1] ^= tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+4 )+(lane4+3)%4+8*__shfl(row,tile4+1)]); if (laneId % 2 == 0) D[3] ^= tmp; else D[0] ^= tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+8 )+(lane4+2)%4+8*__shfl(row,tile4+2)]); if (laneId % 2 == 0) D[2] ^= tmp; else D[3] ^= tmp;
    tmp = __ldg(&S[spacing*2*(16*tile+12)+(lane4+1)%4+8*__shfl(row,tile4+3)]); if (laneId % 2 == 0) D[1] ^= tmp; else D[2] ^= tmp;

    // undo exchange
    if (lane4 >= 2) { __swap(D[0], D[2]); __swap(D[1], D[3]); }

    // undo rotate
    D[1] = __shfl(D[1], rot1);
    D[2] = __shfl(D[2], rot2);
    D[3] = __shfl(D[3], rot3);
}

#define ROTL(a, b) __funnelshift_l( a, a, b );

#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 7); a1^=ROTL(a10, 7); a2^=ROTL(a20, 7); a3^=ROTL(a30, 7);\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 9); a1^=ROTL(a10, 9); a2^=ROTL(a20, 9); a3^=ROTL(a30, 9);\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 13); a1^=ROTL(a10, 13); a2^=ROTL(a20, 13); a3^=ROTL(a30, 13);\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 18); a1^=ROTL(a10, 18); a2^=ROTL(a20, 18); a3^=ROTL(a30, 18);\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
    B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

////////////////////////////////////////////////////////////////////////////////
//! Experimental Scrypt core kernel for Titan devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void test_scrypt_core_kernelA(uint32_t *g_idata)
{
    // add warp specific offsets
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_idata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];

    // registers to store an entire work unit
    uint4 B[4], C[4];

    __transposed_read((uint4*)(g_idata)   , B, 1);
    __transposed_read((uint4*)(g_idata+16), C, 1);

    __transposed_write(B, (uint4*)V, 1024); V+=16;
    __transposed_write(C, (uint4*)V, 1024); V+=16;

    for (int i = 1; i < 1024; i++) {

        xor_salsa8(B, C); xor_salsa8(C, B);

        __transposed_write(B, (uint4*)V, 1024); V+=16;
        __transposed_write(C, (uint4*)V, 1024); V+=16;
    }
}

__global__ void test_scrypt_core_kernelB(uint32_t *g_odata)
{
    // add warp specific offsets
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_odata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];

    // registers to store an entire work unit
    uint4 B[4], C[4];

    __transposed_read((uint4*)(V+1023*32),    B, 1024);
    __transposed_read((uint4*)(V+1023*32+16), C, 1024);

    xor_salsa8(B, C); xor_salsa8(C, B);

    for (int i = 0; i < 1024; i++) {

        __transposed_xor((uint4*)(V),    B, 1024, (C[0].x & 1023));
        __transposed_xor((uint4*)(V+16), C, 1024, (C[0].x & 1023));

        xor_salsa8(B, C); xor_salsa8(C, B);
    }

    __transposed_write(B, (uint4*)(g_odata)   , 1);
    __transposed_write(C, (uint4*)(g_odata+16), 1);
}
