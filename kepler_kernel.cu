#include "hip/hip_runtime.h"
/* Copyright (C) 2013 David G. Andersen. All rights reserved.
 * with modifications by Christian Buchner
 *
 * Use of this code is covered under the Apache 2.0 license, which
 * can be found in the file "LICENSE"
 */

// TODO: attempt V.Volkov style ILP (factor 4)

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#include "miner.h"
#include "kepler_kernel.h"

#define TEXWIDTH 32768

// scratchbuf constants (pointers to scratch buffer for each warp, i.e. 32 hashes)
__constant__ uint32_t* c_V[1024];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1;                  // N-1
// scratch buffer size SCRATCH
__constant__ uint32_t c_SCRATCH;
__constant__ uint32_t c_SCRATCH_WU_PER_WARP;  // SCRATCH * WU_PER_WARP

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

static const int THREADS_PER_SCRYPT_BLOCK = 4;

/* write_keys writes the 8 keys being processed by a warp to the global
 * scratchpad. To effectively use memory bandwidth, it performs the writes
 * (and reads, for read_keys) 128 bytes at a time per memory location
 * by __shfl'ing the 4 entries in bx to the threads in the next-up
 * thread group. It then has eight threads together perform uint4
 * (128 bit) writes to the destination region. This seems to make
 * quite effective use of memory bandwidth. An approach that spread
 * uint32s across more threads was slower because of the increased
 * computation it required.
 *
 * "start" is the loop iteration producing the write - the offset within
 * the block's memory.
 *
 * Internally, this algorithm first __shfl's the 4 bx entries to
 * the next up thread group, and then uses a conditional move to
 * ensure that odd-numbered thread groups exchange the b/bx ordering
 * so that the right parts are written together.
 *
 * Thanks to Babu for helping design the 128-bit-per-write version.
 *
 * _direct lets the caller specify the absolute start location instead of
 * the relative start location, as an attempt to reduce some recomputation.
 */

__device__ __forceinline__
void write_keys_direct(const uint32_t b[4], const uint32_t bx[4], uint32_t start) {

  uint4 t, t2;
  t.x = b[0]; t.y = b[1]; t.z = b[2]; t.w = b[3];

  int target_thread = (threadIdx.x + 4)%32;
  t2.x = __shfl((int)bx[0], target_thread);
  t2.y = __shfl((int)bx[1], target_thread);
  t2.z = __shfl((int)bx[2], target_thread);
  t2.w = __shfl((int)bx[3], target_thread);

  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  uint32_t *scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/(THREADS_PER_SCRYPT_BLOCK * warpSize)];

  unsigned int loc = c ? t2_start : start;
  *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)])) = (c ? t2 : t);
  loc = c ? start : t2_start;
  *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)])) = (c ? t : t2);
}

template <int TEX_DIM> __device__  __forceinline__ void read_keys_direct(uint32_t b[4], uint32_t bx[4], uint32_t start) {

  uint4 t, t2;

  // Tricky bit: We do the work on behalf of thread+4, but then when
  // we steal, we have to steal from (thread+28)%32 to get the right
  // stuff back.
  start = __shfl((int)start, (threadIdx.x & 0x7c)) + 8*(threadIdx.x%4);

  int target_thread = (threadIdx.x + 4)%32;
  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  uint32_t *scratch;
  if (TEX_DIM == 0) scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/(THREADS_PER_SCRYPT_BLOCK * warpSize)];

  unsigned int loc = (c ? t2_start : start) / ((TEX_DIM > 0) ? 4 : 1);
       if (TEX_DIM == 0) t = *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)]));
  else if (TEX_DIM == 1) t = tex1Dfetch(texRef1D_4_V, loc);
  else if (TEX_DIM == 2) t = tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH));
  loc = (c ? start : t2_start) / ((TEX_DIM > 0) ? 4 : 1);;
       if (TEX_DIM == 0) t2 = *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)]));
  else if (TEX_DIM == 1) t2 = tex1Dfetch(texRef1D_4_V, loc/4);
  else if (TEX_DIM == 2) t2 = tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH));

  uint4 tmp = t; t = (c ? t2 : t); t2 = (c ? tmp : t2);
  
  b[0] = t.x; b[1] = t.y; b[2] = t.z; b[3] = t.w;

  int steal_target = (threadIdx.x + 28)%32;

  bx[0] = __shfl((int)t2.x, steal_target);
  bx[1] = __shfl((int)t2.y, steal_target);
  bx[2] = __shfl((int)t2.z, steal_target);
  bx[3] = __shfl((int)t2.w, steal_target);
}


template <int TEX_DIM> __device__  __forceinline__ void read_xor_keys_direct(uint32_t b[4], uint32_t bx[4], uint32_t start) {

  uint4 t, t2;

  // Tricky bit: We do the work on behalf of thread+4, but then when
  // we steal, we have to steal from (thread+28)%32 to get the right
  // stuff back.
  start = __shfl((int)start, (threadIdx.x & 0x7c)) + 8*(threadIdx.x%4);

  int target_thread = (threadIdx.x + 4)%32;
  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  uint32_t *scratch;
  if (TEX_DIM == 0) scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/(THREADS_PER_SCRYPT_BLOCK * warpSize)];

  int loc = (c ? t2_start : start) / ((TEX_DIM > 0) ? 4 : 1);
       if (TEX_DIM == 0) t = *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)]));
  else if (TEX_DIM == 1) t = tex1Dfetch(texRef1D_4_V, loc);
  else if (TEX_DIM == 2) t = tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH));
  loc = (c ? start : t2_start) / ((TEX_DIM > 0) ? 4 : 1);;
       if (TEX_DIM == 0) t2 = *((uint4 *)(&scratch[loc%(c_SCRATCH_WU_PER_WARP)]));
  else if (TEX_DIM == 1) t2 = tex1Dfetch(texRef1D_4_V, loc/4);
  else if (TEX_DIM == 2) t2 = tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH));

  uint4 tmp = t; t = (c ? t2 : t); t2 = (c ? tmp : t2);
  
  b[0] ^= t.x; b[1] ^= t.y; b[2] ^= t.z; b[3] ^= t.w;

  int steal_target = (threadIdx.x + 28)%32;

  bx[0] ^= __shfl((int)t2.x, steal_target);
  bx[1] ^= __shfl((int)t2.y, steal_target);
  bx[2] ^= __shfl((int)t2.z, steal_target);
  bx[3] ^= __shfl((int)t2.w, steal_target);
}


template <int TEX_DIM> __device__  __forceinline__ void read_xor_keys(uint32_t b[4], uint32_t bx[4], uint32_t start) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  start = scrypt_block*c_SCRATCH + (32*start);
  read_xor_keys_direct<TEX_DIM>(b, bx, start);
}


__device__  __forceinline__ void primary_order_shuffle(uint32_t b[4], uint32_t bx[4]) {
  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);
  
  b[3] = __shfl((int)b[3], x1_target_lane);
  b[2] = __shfl((int)b[2], x2_target_lane);
  b[1] = __shfl((int)b[1], x3_target_lane);
  uint32_t tmp = b[1]; b[1] = b[3]; b[3] = tmp;
  
  bx[3] = __shfl((int)bx[3], x1_target_lane);
  bx[2] = __shfl((int)bx[2], x2_target_lane);
  bx[1] = __shfl((int)bx[1], x3_target_lane);
  tmp = bx[1]; bx[1] = bx[3]; bx[3] = tmp;
}

/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__ void load_key_salsa(const uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

  // Read in permuted order. Key loads are not our bottleneck right now.
#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    b[i] = B[key_offset + 4*thread_in_block + (thread_in_block+i)%4];
    bx[i] = B[key_offset + 4*thread_in_block + (thread_in_block+i)%4 + 16];
  }

  primary_order_shuffle(b, bx);
  
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__ void store_key_salsa(uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

  primary_order_shuffle(b, bx);

#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    B[key_offset + 4*thread_in_block + (thread_in_block+i)%4] = b[i];
    B[key_offset + 4*thread_in_block + (thread_in_block+i)%4 + 16] = bx[i];
  }
}


/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__ void load_key_chacha(const uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

  // Read in permuted order. Key loads are not our bottleneck right now.
#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    b[i] = B[key_offset + 4*i + thread_in_block%4];
    bx[i] = B[key_offset + 4*i + thread_in_block%4 + 16];
  }
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__ void store_key_chacha(uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    B[key_offset + 4*i + thread_in_block%4] = b[i];
    B[key_offset + 4*i + thread_in_block%4 + 16] = bx[i];
  }
}


/*
 * salsa_xor_core (Salsa20/8 cypher)
 * The original scrypt called:
 * xor_salsa8(&X[0], &X[16]); <-- the "b" loop
 * xor_salsa8(&X[16], &X[0]); <-- the "bx" loop
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 */

#define XOR_ROTATE_ADD(dst, s1, s2, amt) { uint32_t tmp = x[s1]+x[s2]; x[dst] ^= ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__ void salsa_xor_core(uint32_t b[4], uint32_t bx[4],
                                 const int x1_target_lane,
                                 const int x2_target_lane,
                                 const int x3_target_lane) {
    uint32_t x[4];

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] ^= bx[i];
      x[i] = b[i];
    }

    // Enter in "primary order" (t0 has  0,  4,  8, 12)
    //                          (t1 has  5,  9, 13,  1)
    //                          (t2 has 10, 14,  2,  6)
    //                          (t3 has 15,  3,  7, 11)

#pragma unroll 4
    for (int j = 0; j < 4; j++) {
    
      // Mixing phase of salsa
      XOR_ROTATE_ADD(1, 0, 3, 7);
      XOR_ROTATE_ADD(2, 1, 0, 9);
      XOR_ROTATE_ADD(3, 2, 1, 13);
      XOR_ROTATE_ADD(0, 3, 2, 18);
      
      /* Transpose rows and columns. */
      /* Unclear if this optimization is needed: These are ordered based
       * upon the dependencies needed in the later xors. Compiler should be
       * able to figure this out, but might as well give it a hand. */
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      
      /* The next XOR_ROTATE_ADDS could be written to be a copy-paste of the first,
       * but the register targets are rewritten here to swap x[1] and x[3] so that
       * they can be directly shuffled to and from our peer threads without
       * reassignment. The reverse shuffle then puts them back in the right place.
       */
      
      XOR_ROTATE_ADD(3, 0, 1, 7);
      XOR_ROTATE_ADD(2, 3, 0, 9);
      XOR_ROTATE_ADD(1, 2, 3, 13);
      XOR_ROTATE_ADD(0, 1, 2, 18);
      
      x[3] = __shfl((int)x[3], x3_target_lane);
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
    }

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] += x[i];
      // The next two lines are the beginning of the BX-centric loop iteration
      bx[i] ^= b[i];
      x[i] = bx[i];
    }

    // This is a copy of the same loop above, identical but stripped of comments.
    // Duplicated so that we can complete a bx-based loop with fewer register moves.
#pragma unroll 4
    for (int j = 0; j < 4; j++) {
      XOR_ROTATE_ADD(1, 0, 3, 7);
      XOR_ROTATE_ADD(2, 1, 0, 9);
      XOR_ROTATE_ADD(3, 2, 1, 13);
      XOR_ROTATE_ADD(0, 3, 2, 18);
      
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      
      XOR_ROTATE_ADD(3, 0, 1, 7);
      XOR_ROTATE_ADD(2, 3, 0, 9);
      XOR_ROTATE_ADD(1, 2, 3, 13);
      XOR_ROTATE_ADD(0, 1, 2, 18);
      
      x[3] = __shfl((int)x[3], x3_target_lane);
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
    }

    // At the end of these iterations, the data is in primary order again.
#undef XOR_ROTATE_ADD

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      bx[i] += x[i];
    }
}


/*
 * chacha_xor_core (ChaCha20/8 cypher)
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 * 
 * load_key and store_key must not use primary order when
 * using ChaCha20/8, but rather the basic transposed order
 * (referred to as "column mode" below)
 */

#define CHACHA_PRIMITIVE(pt, rt, ps, amt) { uint32_t tmp = x[rt] ^ (x[pt] += x[ps]); x[rt] = ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__ void chacha_xor_core(uint32_t b[4], uint32_t bx[4],
                                 const int x1_target_lane,
                                 const int x2_target_lane,
                                 const int x3_target_lane) {
    uint32_t x[4];

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] ^= bx[i];
      x[i] = b[i];
    }

    // Enter in "column" mode (t0 has 0, 4,  8, 12)
    //                        (t1 has 1, 5,  9, 13)
    //                        (t2 has 2, 6, 10, 14)
    //                        (t3 has 3, 7, 11, 15)

#pragma unroll 4
    for (int j = 0; j < 4; j++) {
    
      // Column Mixing phase of chacha
      CHACHA_PRIMITIVE(0 ,3, 1, 16)
      CHACHA_PRIMITIVE(2 ,1, 3, 12)
      CHACHA_PRIMITIVE(0 ,3, 1,  8)
      CHACHA_PRIMITIVE(2 ,1, 3,  7)
      
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      x[3] = __shfl((int)x[3], x3_target_lane);
      
      // Diagonal Mixing phase of chacha
      CHACHA_PRIMITIVE(0 ,3, 1, 16)
      CHACHA_PRIMITIVE(2 ,1, 3, 12)
      CHACHA_PRIMITIVE(0 ,3, 1,  8)
      CHACHA_PRIMITIVE(2 ,1, 3,  7)
      
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
    }

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] += x[i];
      // The next two lines are the beginning of the BX-centric loop iteration
      bx[i] ^= b[i];
      x[i] = bx[i];
    }

#pragma unroll 4
    for (int j = 0; j < 4; j++) {

      // Column Mixing phase of chacha
      CHACHA_PRIMITIVE(0 ,3, 1, 16)
      CHACHA_PRIMITIVE(2 ,1, 3, 12)
      CHACHA_PRIMITIVE(0 ,3, 1,  8)
      CHACHA_PRIMITIVE(2 ,1, 3,  7)
      
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      x[3] = __shfl((int)x[3], x3_target_lane);
      
      // Diagonal Mixing phase of chacha
      CHACHA_PRIMITIVE(0 ,3, 1, 16)
      CHACHA_PRIMITIVE(2 ,1, 3, 12)
      CHACHA_PRIMITIVE(0 ,3, 1,  8)
      CHACHA_PRIMITIVE(2 ,1, 3,  7)
      
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
    }

#undef CHACHA_PRIMITIVE

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      bx[i] += x[i];
    }
}


/*
 * The hasher_gen_kernel operates on a group of 1024-bit input keys
 * in B, stored as:
 * B = { k1B k1Bx k2B k2Bx ... }
 * and fills up the scratchpad with the iterative hashes derived from
 * those keys:
 * scratch { k1h1B k1h1Bx K1h2B K1h2Bx ... K2h1B K2h1Bx K2h2B K2h2Bx ... }
 * scratch is 1024 times larger than the input keys B.
 * It is extremely important to stream writes effectively into scratch;
 * less important to coalesce the reads from B.
 *
 * Key ordering note: Keys are input from B in "original" order:
 * K = {k1, k2, k3, k4, k5, ..., kx15, kx16, kx17, ..., kx31 }
 * After inputting into kernel_gen, each component k and kx of the
 * key is transmuted into a permuted internal order to make processing faster:
 * K = k, kx with:
 * k = 0, 4, 8, 12, 5, 9, 13, 1, 10, 14, 2, 6, 15, 3, 7, 11
 * and similarly for kx.
 */

template <int ALGO> __global__
void kepler_scrypt_core_kernelA(const uint32_t *d_idata, int begin, int end) {

  /* Each thread operates on four of the sixteen B and Bx variables. Thus,
   * each key is processed by four threads in parallel. salsa_scrypt_core
   * internally shuffles the variables between threads (and back) as
   * needed.
   */
  uint32_t b[4], bx[4];

  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);

  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int start = scrypt_block*c_SCRATCH + 8*(threadIdx.x%4);

  int i=begin;

  if (i == 0)
  {
      switch(ALGO) {
        case ALGO_SCRYPT:      load_key_salsa(d_idata, b, bx); break;
        case ALGO_SCRYPT_JANE: load_key_chacha(d_idata, b, bx); break;
      }
      write_keys_direct(b, bx, start);
      ++i;
  }
  else
  {
      read_keys_direct<0>(b, bx, start+32*(i-1));
  }
  
  while (i < end) {
    switch(ALGO) {
      case ALGO_SCRYPT:      salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
      case ALGO_SCRYPT_JANE: chacha_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
    }
    write_keys_direct(b, bx, start+32*i);
    ++i;
  }
}


/*
 * hasher_hash_kernel runs the second phase of scrypt after the scratch
 * buffer is filled with the iterative hashes: It bounces through
 * the scratch buffer in pseudorandom order, mixing the key as it goes.
 */

template <int ALGO, int TEX_DIM> __global__
void kepler_scrypt_core_kernelB(uint32_t *d_odata, int begin, int end) {

  /* Each thread operates on a group of four variables that must be processed
   * together. Shuffle between threaads in a warp between iterations.
   */
  uint32_t b[4], bx[4];

  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int start = scrypt_block*c_SCRATCH + 8*(threadIdx.x%4);

  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);

  if (begin == 0)
  {
      read_keys_direct<TEX_DIM>(b, bx, start+32*(c_N_1));

      switch(ALGO) {
        case ALGO_SCRYPT:      salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
        case ALGO_SCRYPT_JANE: chacha_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
      }
  }
  else
  {
      switch(ALGO) {
        case ALGO_SCRYPT:      load_key_salsa(d_odata, b, bx); break;
        case ALGO_SCRYPT_JANE: load_key_chacha(d_odata, b, bx); break;
      }
  }

  for (int i = begin; i < end; i++) {

    // Bounce through the key space and XOR the new keys in.
    // Critical thing: (X[16] & (c_N_1)) tells us the next slot to read.
    // X[16] in the original is bx[0]
    int slot = bx[0] & (c_N_1);
    read_xor_keys<TEX_DIM>(b, bx, slot);
    switch(ALGO) {
      case ALGO_SCRYPT:      salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
      case ALGO_SCRYPT_JANE: chacha_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane); break;
    }
  }

  switch(ALGO) {
    case ALGO_SCRYPT:      store_key_salsa(d_odata, b, bx); break;
    case ALGO_SCRYPT_JANE: store_key_chacha(d_odata, b, bx); break;
  }
  
}

KeplerKernel::KeplerKernel() : KernelInterface()
{
}

bool KeplerKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef1D_4_V.normalized = 0;
    texRef1D_4_V.filterMode = hipFilterModePoint;
    texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
    return true;
}

bool KeplerKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef2D_4_V.normalized = 0;
    texRef2D_4_V.filterMode = hipFilterModePoint;
    texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
    texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
    // maintain texture width of TEXWIDTH (max. limit is 65000)
    while (width > TEXWIDTH) { width /= 2; height *= 2; pitch /= 2; }
    while (width < TEXWIDTH) { width *= 2; height = (height+1)/2; pitch *= 2; }
//    fprintf(stderr, "total size: %u, %u bytes\n", pitch * height, width * sizeof(uint32_t) * 4 * height);
//    fprintf(stderr, "binding width width=%d, height=%d, pitch=%d\n", width, height,pitch);
    checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
    return true;
}

bool KeplerKernel::unbindtexture_1D()
{
    checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
    return true;
}

bool KeplerKernel::unbindtexture_2D()
{
    checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
    return true;
}

void KeplerKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool KeplerKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // this kernel needs 4 threads per work unit. 
    switch(opt_algo) {
      case ALGO_SCRYPT: grid.x *= 4; break; // scrypt: We scale up the grid x dimension to compensate.
      case ALGO_SCRYPT_JANE: threads.x *= 4; break; // scrypt-jane: we scale up thread block size
    }
    
    // make some constants available to kernel, update only initially and when changing
    static int prev_N = 0;
    if (N != prev_N) {
        uint32_t h_N = N;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        prev_N = N;
        uint32_t h_N_1 = N-1;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        uint32_t h_SCRATCH = SCRATCH;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH), &h_SCRATCH, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        uint32_t h_SCRATCH_WU_PER_WARP = SCRATCH * WU_PER_WARP;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH_WU_PER_WARP), &h_SCRATCH_WU_PER_WARP, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
    }

    // First phase: Sequential writes to scratchpad.

    int batch = device_batchsize[thr_id];

    int pos = 0;
    do 
    {
        switch(opt_algo) {
          case ALGO_SCRYPT: kepler_scrypt_core_kernelA<ALGO_SCRYPT><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N)); break;
          case ALGO_SCRYPT_JANE: kepler_scrypt_core_kernelA<ALGO_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N)); break;
        }

        // Optional millisecond sleep in between kernels

        if (!benchmark && interactive) {
            checkCudaErrors(MyStreamSynchronize(stream, -1, thr_id));
            usleep(100);
        }
        pos += batch;
    } while (pos < N);

    // Second phase: Random read access from scratchpad.

    pos = 0;
    do
    {
        if (pos > 0 && !benchmark && interactive) {
            checkCudaErrors(MyStreamSynchronize(stream, -1, thr_id));
            usleep(100);
        }

        if (texture_cache)
        {
            if (texture_cache == 1)
                switch(opt_algo) {
                    case ALGO_SCRYPT: kepler_scrypt_core_kernelB<ALGO_SCRYPT,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                    case ALGO_SCRYPT_JANE: kepler_scrypt_core_kernelB<ALGO_SCRYPT_JANE,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                }
            else if (texture_cache == 2)
                switch(opt_algo) {
                    case ALGO_SCRYPT: kepler_scrypt_core_kernelB<ALGO_SCRYPT,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                    case ALGO_SCRYPT_JANE: kepler_scrypt_core_kernelB<ALGO_SCRYPT_JANE,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                }
        }
        else
            switch(opt_algo) {
                case ALGO_SCRYPT: kepler_scrypt_core_kernelB<ALGO_SCRYPT,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                case ALGO_SCRYPT_JANE: kepler_scrypt_core_kernelB<ALGO_SCRYPT_JANE,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
        }
        pos += batch;
    } while (pos < N);

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}
