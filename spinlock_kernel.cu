#include "hip/hip_runtime.h"
//
// Kernel that runs best on Kepler (Compute 3.0) devices
//
// - makes use of 8 byte of Kepler's shared memory bank mode
// - does memory transfers with ulonglong2 vectors whereever possible
// - further halves shared memory consumption over Fermi kernel by sharing
//   the same shared memory buffers among two neighbor warps. Requires spinlocks
//   based on global atomics and hence Compute 1.1
//  
// NOTE: compile this .cu module for compute_11,sm_11 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "spinlock_kernel.h"

// forward references
template <int WARPS_PER_BLOCK> __global__ void spinlock_scrypt_core_kernelA(uint32_t *g_idata, int *mutex);
template <int WARPS_PER_BLOCK> __global__ void spinlock_scrypt_core_kernelB(uint32_t *g_odata, int *mutex);
template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void spinlock_scrypt_core_kernelB_tex(uint32_t *g_odata, int *mutex);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

SpinlockKernel::SpinlockKernel() : KernelInterface()
{
}

bool SpinlockKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef1D_4_V.normalized = 0;
    texRef1D_4_V.filterMode = hipFilterModePoint;
    texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
    return true;
}

bool SpinlockKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
    texRef2D_4_V.normalized = 0;
    texRef2D_4_V.filterMode = hipFilterModePoint;
    texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
    texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
    return true;
}

bool SpinlockKernel::unbindtexture_1D()
{
    checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
    return true;
}

bool SpinlockKernel::unbindtexture_2D()
{
    checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
    return true;
}

void SpinlockKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool SpinlockKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: spinlock_scrypt_core_kernelA<1><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 2: spinlock_scrypt_core_kernelA<2><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 3: spinlock_scrypt_core_kernelA<3><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 4: spinlock_scrypt_core_kernelA<4><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 5: spinlock_scrypt_core_kernelA<5><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 6: spinlock_scrypt_core_kernelA<6><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 7: spinlock_scrypt_core_kernelA<7><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 8: spinlock_scrypt_core_kernelA<8><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 9: spinlock_scrypt_core_kernelA<9><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 10: spinlock_scrypt_core_kernelA<10><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 11: spinlock_scrypt_core_kernelA<11><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 12: spinlock_scrypt_core_kernelA<12><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 13: spinlock_scrypt_core_kernelA<13><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 14: spinlock_scrypt_core_kernelA<14><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 15: spinlock_scrypt_core_kernelA<15><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 16: spinlock_scrypt_core_kernelA<16><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 17: spinlock_scrypt_core_kernelA<17><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 18: spinlock_scrypt_core_kernelA<18><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 19: spinlock_scrypt_core_kernelA<19><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 20: spinlock_scrypt_core_kernelA<20><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 21: spinlock_scrypt_core_kernelA<21><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 22: spinlock_scrypt_core_kernelA<22><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 23: spinlock_scrypt_core_kernelA<23><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 24: spinlock_scrypt_core_kernelA<24><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        default: success = false; break;
    }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    if (texture_cache)
    {
        if (texture_cache == 1)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: spinlock_scrypt_core_kernelB_tex<1,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 2: spinlock_scrypt_core_kernelB_tex<2,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 3: spinlock_scrypt_core_kernelB_tex<3,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 4: spinlock_scrypt_core_kernelB_tex<4,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 5: spinlock_scrypt_core_kernelB_tex<5,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 6: spinlock_scrypt_core_kernelB_tex<6,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 7: spinlock_scrypt_core_kernelB_tex<7,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 8: spinlock_scrypt_core_kernelB_tex<8,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 9: spinlock_scrypt_core_kernelB_tex<9,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 10: spinlock_scrypt_core_kernelB_tex<10,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 11: spinlock_scrypt_core_kernelB_tex<11,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 12: spinlock_scrypt_core_kernelB_tex<12,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 13: spinlock_scrypt_core_kernelB_tex<13,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 14: spinlock_scrypt_core_kernelB_tex<14,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 15: spinlock_scrypt_core_kernelB_tex<15,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 16: spinlock_scrypt_core_kernelB_tex<16,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 17: spinlock_scrypt_core_kernelB_tex<17,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 18: spinlock_scrypt_core_kernelB_tex<18,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 19: spinlock_scrypt_core_kernelB_tex<19,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 20: spinlock_scrypt_core_kernelB_tex<20,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 21: spinlock_scrypt_core_kernelB_tex<21,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 22: spinlock_scrypt_core_kernelB_tex<22,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 23: spinlock_scrypt_core_kernelB_tex<23,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 24: spinlock_scrypt_core_kernelB_tex<24,1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                default: success = false; break;
            }
        }
        else if (texture_cache == 2)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: spinlock_scrypt_core_kernelB_tex<1,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 2: spinlock_scrypt_core_kernelB_tex<2,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 3: spinlock_scrypt_core_kernelB_tex<3,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 4: spinlock_scrypt_core_kernelB_tex<4,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 5: spinlock_scrypt_core_kernelB_tex<5,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 6: spinlock_scrypt_core_kernelB_tex<6,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 7: spinlock_scrypt_core_kernelB_tex<7,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 8: spinlock_scrypt_core_kernelB_tex<8,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 9: spinlock_scrypt_core_kernelB_tex<9,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 10: spinlock_scrypt_core_kernelB_tex<10,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 11: spinlock_scrypt_core_kernelB_tex<11,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 12: spinlock_scrypt_core_kernelB_tex<12,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 13: spinlock_scrypt_core_kernelB_tex<13,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 14: spinlock_scrypt_core_kernelB_tex<14,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 15: spinlock_scrypt_core_kernelB_tex<15,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 16: spinlock_scrypt_core_kernelB_tex<16,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 17: spinlock_scrypt_core_kernelB_tex<17,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 18: spinlock_scrypt_core_kernelB_tex<18,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 19: spinlock_scrypt_core_kernelB_tex<19,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 20: spinlock_scrypt_core_kernelB_tex<20,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 21: spinlock_scrypt_core_kernelB_tex<21,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 22: spinlock_scrypt_core_kernelB_tex<22,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 23: spinlock_scrypt_core_kernelB_tex<23,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                case 24: spinlock_scrypt_core_kernelB_tex<24,2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
                default: success = false; break;
            }
        } else success = false;
    }
    else
    {
        switch (WARPS_PER_BLOCK) {
            case 1: spinlock_scrypt_core_kernelB<1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 2: spinlock_scrypt_core_kernelB<2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 3: spinlock_scrypt_core_kernelB<3><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 4: spinlock_scrypt_core_kernelB<4><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 5: spinlock_scrypt_core_kernelB<5><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 6: spinlock_scrypt_core_kernelB<6><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 7: spinlock_scrypt_core_kernelB<7><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 8: spinlock_scrypt_core_kernelB<8><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 9: spinlock_scrypt_core_kernelB<9><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 10: spinlock_scrypt_core_kernelB<10><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 11: spinlock_scrypt_core_kernelB<11><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 12: spinlock_scrypt_core_kernelB<12><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 13: spinlock_scrypt_core_kernelB<13><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 14: spinlock_scrypt_core_kernelB<14><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 15: spinlock_scrypt_core_kernelB<15><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 16: spinlock_scrypt_core_kernelB<16><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 17: spinlock_scrypt_core_kernelB<17><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 18: spinlock_scrypt_core_kernelB<18><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 19: spinlock_scrypt_core_kernelB<19><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 20: spinlock_scrypt_core_kernelB<20><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 21: spinlock_scrypt_core_kernelB<21><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 22: spinlock_scrypt_core_kernelB<22><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 23: spinlock_scrypt_core_kernelB<23><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            case 24: spinlock_scrypt_core_kernelB<24><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
            default: success = false; break;
        }
    }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<7) | ((a00)>>25) );\
a1^=(((a10)<<7) | ((a10)>>25) );\
a2^=(((a20)<<7) | ((a20)>>25) );\
a3^=(((a30)<<7) | ((a30)>>25) );\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<9) | ((a00)>>23) );\
a1^=(((a10)<<9) | ((a10)>>23) );\
a2^=(((a20)<<9) | ((a20)>>23) );\
a3^=(((a30)<<9) | ((a30)>>23) );\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<13) | ((a00)>>19) );\
a1^=(((a10)<<13) | ((a10)>>19) );\
a2^=(((a20)<<13) | ((a20)>>19) );\
a3^=(((a30)<<13) | ((a30)>>19) );\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<18) | ((a00)>>14) );\
a1^=(((a10)<<18) | ((a10)>>14) );\
a2^=(((a20)<<18) | ((a20)>>14) );\
a3^=(((a30)<<18) | ((a30)>>14) );\
};\

static __device__ void xor_salsa8(uint32_t *B,uint32_t *C)
{
	uint32_t x[16];
	x[0]=(B[0] ^= C[0]);
	x[1]=(B[1] ^= C[1]);
	x[2]=(B[2] ^= C[2]);
	x[3]=(B[3] ^= C[3]);
	x[4]=(B[4] ^= C[4]);
	x[5]=(B[5] ^= C[5]);
	x[6]=(B[6] ^= C[6]);
	x[7]=(B[7] ^= C[7]);
	x[8]=(B[8] ^= C[8]);
	x[9]=(B[9] ^= C[9]);
	x[10]=(B[10] ^= C[10]);
	x[11]=(B[11] ^= C[11]);
	x[12]=(B[12] ^= C[12]);
	x[13]=(B[13] ^= C[13]);
	x[14]=(B[14] ^= C[14]);
	x[15]=(B[15] ^= C[15]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    B[ 0] += x[0]; B[ 1] += x[1]; B[ 2] += x[2]; B[ 3] += x[3]; B[ 4] += x[4]; B[ 5] += x[5]; B[ 6] += x[6]; B[ 7] += x[7];
    B[ 8] += x[8]; B[ 9] += x[9]; B[10] += x[10]; B[11] += x[11]; B[12] += x[12]; B[13] += x[13]; B[14] += x[14]; B[15] += x[15];
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    left.z ^= right.z;
    left.w ^= right.w;
    return left;
}

static __device__ ulonglong2& operator^=(ulonglong2& left, const ulonglong2& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    return left;
}

static __device__ void lock(int *mutex, int i)
{
    while( atomicCAS( &mutex[i], 0, 1 ) != 0 )
    {
        // keep the (slow) special function unit busy to avoid hammering
        // the memory controller with atomic operations while busy waiting
        asm volatile("{\t\n.reg .f32 tmp;\t\n"
                     "lg2.approx.f32 tmp, 0f00000000;\t\n"
                     "lg2.approx.f32 tmp, 0f00000000;\t\n}" :: );
    }
}

static __device__ void unlock(int *mutex, int i)
{
    atomicExch( &mutex[i], 0 );
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel using spinlocks to cut shared memory use in half.
//! Ideal for Kepler devices where shared memory use prevented optimal occupancy.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
spinlock_scrypt_core_kernelA(uint32_t *g_idata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+2];

    int warpIdx        = threadIdx.x / warpSize;
    int warpThread     = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/4;
    unsigned int Z = 4*(warpThread%4);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP]  + SCRATCH*Y + Z;

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    volatile int warpIdx_2      = warpIdx/2;
    uint32_t ((*XB)[16+2]) = (uint32_t (*)[16+2])&X[warpIdx_2][Y][Z];
    uint32_t *XX = X[warpIdx_2][warpThread];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&V[SCRATCH*wu])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = XX[idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&V[SCRATCH*wu+16])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = XX[idx];

    for (int i = 1; i < 1024; i++) {

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((ulonglong2*)(&V[SCRATCH*wu + i*32])) = *((ulonglong2*)XB[wu]);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((ulonglong2*)(&V[SCRATCH*wu + i*32 + 16])) = *((ulonglong2*)XB[wu]);
    }
    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
}

template <int WARPS_PER_BLOCK> __global__ void
spinlock_scrypt_core_kernelB(uint32_t *g_odata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+2];

    int warpIdx        = threadIdx.x / warpSize;
    int warpThread     = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/4;
    unsigned int Z = 4*(warpThread%4);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    volatile int warpIdx_2      = warpIdx/2;
    uint32_t ((*XB)[16+2]) = (uint32_t (*)[16+2])&X[warpIdx_2][Y][Z];
    uint32_t *XX = X[warpIdx_2][warpThread];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + 1023*32]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = XX[idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + 1023*32 + 16]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = XX[idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

    for (int i = 0; i < 1024; i++) {

        XX[16] = 32 * (C[0] & 1023);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((ulonglong2*)XB[wu]) ^= *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z]]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) B[idx] = XX[idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((ulonglong2*)XB[wu]) ^= *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z] + 16]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) C[idx] = XX[idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) XX[idx] = B[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) XX[idx] = C[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
}

template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void
spinlock_scrypt_core_kernelB_tex(uint32_t *g_odata, int *mutex)
{
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+2];

    int warpIdx        = threadIdx.x / warpSize;
    int warpThread     = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/4;
    unsigned int Z = 4*(warpThread%4);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    volatile int warpIdx_2      = warpIdx/2;
    uint32_t ((*XB)[16+2]) = (uint32_t (*)[16+2])&X[warpIdx_2][Y][Z];
    uint32_t *XX = X[warpIdx_2][warpThread];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
                    tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 1023*32 + Z)/4) :
                    tex2D(texRef2D_4_V, 0.5f + (32*1023 + Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = XX[idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
                    tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + 1023*32 + 16+Z)/4) :
                    tex2D(texRef2D_4_V, 0.5f + (32*1023 + 16+Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = XX[idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

    for (int i = 0; i < 1024; i++) {

        XX[16] = 32 * (C[0] & 1023);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)XB[wu]) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + Z)/4) :
                        tex2D(texRef2D_4_V, 0.5f + (XB[wu][16-Z] + Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) B[idx] = XX[idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) XX[idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)XB[wu]) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_4_V, (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + 16+Z)/4) :
                        tex2D(texRef2D_4_V, 0.5f + (XB[wu][16-Z] + 16+Z)/4, 0.5f + (offset+wu+Y)));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) C[idx] = XX[idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) XX[idx] = B[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) XX[idx] = C[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
}
