#include "hip/hip_runtime.h"
//
// Kernel that runs best on Kepler (Compute 3.5) devices
// uses funnel shifter and __ldg() intrinsic, but suffers from unfavorable
// shared memory alignment (+4 instead of +1) due to different PTX ISA
//
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "titan_kernel.h"

// forward references
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernel_titanA(uint32_t *g_idata, int *mutex);
template <int WARPS_PER_BLOCK> __global__ void scrypt_core_kernel_titanB(uint32_t *g_odata, int *mutex);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

TitanKernel::TitanKernel() : KernelInterface()
{
    hipDeviceSetSharedMemConfig ( hipSharedMemBankSizeEightByte );
}

void TitanKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool TitanKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, int *mutex, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: scrypt_core_kernel_titanA<1><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 2: scrypt_core_kernel_titanA<2><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 3: scrypt_core_kernel_titanA<3><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 4: scrypt_core_kernel_titanA<4><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 5: scrypt_core_kernel_titanA<5><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 6: scrypt_core_kernel_titanA<6><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 7: scrypt_core_kernel_titanA<7><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 8: scrypt_core_kernel_titanA<8><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 9: scrypt_core_kernel_titanA<9><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 10: scrypt_core_kernel_titanA<10><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 11: scrypt_core_kernel_titanA<11><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 12: scrypt_core_kernel_titanA<12><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 13: scrypt_core_kernel_titanA<13><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 14: scrypt_core_kernel_titanA<14><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 15: scrypt_core_kernel_titanA<15><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        case 16: scrypt_core_kernel_titanA<16><<< grid, threads, 0, stream >>>(d_idata, mutex); break;
        default: success = false; break;
    }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: scrypt_core_kernel_titanB<1><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 2: scrypt_core_kernel_titanB<2><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 3: scrypt_core_kernel_titanB<3><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 4: scrypt_core_kernel_titanB<4><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 5: scrypt_core_kernel_titanB<5><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 6: scrypt_core_kernel_titanB<6><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 7: scrypt_core_kernel_titanB<7><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 8: scrypt_core_kernel_titanB<8><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 9: scrypt_core_kernel_titanB<9><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 10: scrypt_core_kernel_titanB<10><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 11: scrypt_core_kernel_titanB<11><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 12: scrypt_core_kernel_titanB<12><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 13: scrypt_core_kernel_titanB<13><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 14: scrypt_core_kernel_titanB<14><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 15: scrypt_core_kernel_titanB<15><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        case 16: scrypt_core_kernel_titanB<16><<< grid, threads, 0, stream >>>(d_odata, mutex); break;
        default: success = false; break;
    }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL(a, b) __funnelshift_l( a, a, b );

static __device__  void xor_salsa8(uint32_t* __restrict__ B, const uint32_t* __restrict__ C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

static __device__ __forceinline__ uint4& operator^=(uint4& left, const uint4& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    left.z ^= right.z;
    left.w ^= right.w;
    return left;
}

static __device__ __forceinline__ void lock(int *mutex, int i)
{
    while( atomicCAS( &mutex[i], 0, 1 ) != 0 )
    {
        // keep the (slow) special function unit busy to avoid hammering
        // the memory controller with atomic operations while busy waiting
        asm volatile("{\t\n.reg .f32 tmp;\t\n"
                     "lg2.approx.f32 tmp, 0f00000000;\t\n"
                     "lg2.approx.f32 tmp, 0f00000000;\t\n}" :: );
    }
}

static __device__ __forceinline__ void unlock(int *mutex, int i)
{
    atomicExch( &mutex[i], 0 );
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel using titans to cut shared memory use in half.
//! Ideal for Kepler devices where shared memory use prevented optimal occupancy.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_titanA(uint32_t *g_idata, int *mutex)
{
     // bank conflict mitigation:  +4 for alignment for uint4 in PTX >=2.0 ISA
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+4];

    volatile int warpIdx        = threadIdx.x / warpSize;
    volatile int warpThread     = threadIdx.x % warpSize;
    volatile int warpIdx_2      = warpIdx/2;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/4;
    unsigned int Z = 4*(warpThread%4);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32 * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&V[SCRATCH*wu])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z])) = *((uint4*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx_2][warpThread][idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&V[SCRATCH*wu+16])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z])) = *((uint4*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx_2][warpThread][idx];

    for (int i = 1; i < 1024; i++) {

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&V[SCRATCH*wu + i*32])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z]));

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&V[SCRATCH*wu + i*32 + 16])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z]));
    }
    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
}

template <int WARPS_PER_BLOCK> __global__ void
scrypt_core_kernel_titanB(uint32_t *g_odata, int *mutex)
{
    // bank conflict mitigation:  +4 for alignment for uint4 in PTX >=2.0 ISA
    __shared__ uint32_t X[(WARPS_PER_BLOCK+1)/2][WU_PER_WARP][16+4];

    volatile int warpIdx        = threadIdx.x / warpSize;
    volatile int warpThread     = threadIdx.x % warpSize;
    volatile int warpIdx_2      = warpIdx/2;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/4;
    unsigned int Z = 4*(warpThread%4);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32 * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

    // registers to store an entire work unit
    uint32_t B[16], C[16];

    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx_2][wu+Y][Z])) = __ldg((uint4*)(&V[SCRATCH*wu + 1023*32]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx_2][warpThread][idx];

#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&X[warpIdx_2][wu+Y][Z])) = __ldg((uint4*)(&V[SCRATCH*wu + 1023*32 + 16]));
#pragma unroll 16
    for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx_2][warpThread][idx];

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    xor_salsa8(B, C); xor_salsa8(C, B);
    if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);

    for (int i = 0; i < 1024; i++) {

        X[warpIdx_2][warpThread][16] = C[0];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = B[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx_2][wu+Y][Z])) ^= __ldg((uint4*)(&V[SCRATCH*wu + 32*(X[warpIdx_2][wu+Y][16] & 1023)]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) B[idx] = X[warpIdx_2][warpThread][idx];

#pragma unroll 16
        for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = C[idx];
#pragma unroll 4
        for (int wu=0; wu < 32; wu+=8)
            *((uint4*)(&X[warpIdx_2][wu+Y][Z])) ^= __ldg((uint4*)(&V[SCRATCH*wu + 32*(X[warpIdx_2][wu+Y][16] & 1023) + 16]));
#pragma unroll 16
        for (int idx=0; idx < 16; idx++) C[idx] = X[warpIdx_2][warpThread][idx];

        if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
        xor_salsa8(B, C); xor_salsa8(C, B);
        if (warpThread == 0) lock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
    }

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = B[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+Z])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z]));

#pragma unroll 16
    for (int idx=0; idx < 16; ++idx) X[warpIdx_2][warpThread][idx] = C[idx];
#pragma unroll 4
    for (int wu=0; wu < 32; wu+=8)
        *((uint4*)(&g_odata[32*(wu+Y)+16+Z])) = *((uint4*)(&X[warpIdx_2][wu+Y][Z]));

    if (warpThread == 0) unlock(mutex, blockIdx.x * (WARPS_PER_BLOCK+1)/2 + warpIdx_2);
}
