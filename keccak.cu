#include "hip/hip_runtime.h"
//
//  =============== KECCAK part on nVidia GPU ======================
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//
// TODO: the actual CUDA porting work is work in progress...
//
//       For good performance we have to get rid of most local memory spills
//       TODO: drop the uint8_t conversions (use uint32_t arrays or uint64_t)
//             manually inline scrypt_pbkdf2_1 function into pre/post kernels
//             and unroll loops. Remove unused code.

#include <map>
#include <stdint.h>

#include "salsa_kernel.h"
#include "miner.h"

#include "keccak.h"

// define some error checking macros
#undef checkCudaErrors

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
    { \
        applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) calling '%s' (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), #x, __FILE__, __LINE__); \
    } \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

#define U8TO32_BE(p)                                            \
	(((uint32_t)((p)[0]) << 24) | ((uint32_t)((p)[1]) << 16) |  \
	 ((uint32_t)((p)[2]) <<  8) | ((uint32_t)((p)[3])      ))

#define U8TO32_LE(p)                                            \
	(((uint32_t)((p)[0])      ) | ((uint32_t)((p)[1]) <<  8) |  \
	 ((uint32_t)((p)[2]) << 16) | ((uint32_t)((p)[3]) << 24))

#define U32TO8_BE(p, v)                                           \
	(p)[0] = (uint8_t)((v) >> 24); (p)[1] = (uint8_t)((v) >> 16); \
	(p)[2] = (uint8_t)((v) >>  8); (p)[3] = (uint8_t)((v)      );

#define U32TO8_LE(p, v)                                           \
	(p)[0] = (uint8_t)((v)      ); (p)[1] = (uint8_t)((v) >>  8); \
	(p)[2] = (uint8_t)((v) >> 16); (p)[3] = (uint8_t)((v) >> 24);

#define U8TO64_BE(p)                                                  \
	(((uint64_t)U8TO32_BE(p) << 32) | (uint64_t)U8TO32_BE((p) + 4))

#define U8TO64_LE(p)                                                  \
	(((uint64_t)U8TO32_LE(p)) | ((uint64_t)U8TO32_LE((p) + 4) << 32))

#define U64TO8_BE(p, v)                        \
	U32TO8_BE((p),     (uint32_t)((v) >> 32)); \
	U32TO8_BE((p) + 4, (uint32_t)((v)      ));

#define U64TO8_LE(p, v)                        \
	U32TO8_LE((p),     (uint32_t)((v)      )); \
	U32TO8_LE((p) + 4, (uint32_t)((v) >> 32));

#define U32_SWAP(v) {                                             \
	(v) = (((v) << 8) & 0xFF00FF00 ) | (((v) >> 8) & 0xFF00FF );  \
    (v) = ((v) << 16) | ((v) >> 16);                              \
}

#define U64_SWAP(v) {                                                                       \
	(v) = (((v) <<  8) & 0xFF00FF00FF00FF00ull ) | (((v) >>  8) & 0x00FF00FF00FF00FFull );  \
	(v) = (((v) << 16) & 0xFFFF0000FFFF0000ull ) | (((v) >> 16) & 0x0000FFFF0000FFFFull );  \
    (v) = ((v) << 32) | ((v) >> 32);                                                        \
}

#define ROTL64(a,b) (((a) << (b)) | ((a) >> (64 - b)))

// ---------------------------- BEGIN keccak functions ------------------------------------

#define KECCAK_HASH "Keccak-512"
#define KECCAK_HASH_DIGEST_SIZE 64
#define KECCAK_F 1600
#define KECCAK_C (KECCAK_HASH_DIGEST_SIZE * 8 * 2) /* 1024 */
#define KECCAK_R (KECCAK_F - KECCAK_C) /* 576 */
#define KECCAK_HASH_BLOCK_SIZE (KECCAK_R / 8)

typedef uint8_t keccak_hash_digest[KECCAK_HASH_DIGEST_SIZE];

typedef struct keccak_hash_state_t {
	uint64_t state[KECCAK_F / 64]; // 25
	uint32_t leftover;
	uint8_t buffer[KECCAK_HASH_BLOCK_SIZE]; // 72
} keccak_hash_state;

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];
__constant__ uint32_t pdata[20];

__device__ void
keccak_block(keccak_hash_state *S, const uint8_t *in) {
	size_t i;
	uint64_t *s = S->state, t[5], u[5], v, w;

	/* absorb input */
	for (i = 0; i < KECCAK_HASH_BLOCK_SIZE / 8; i++, in += 8)
		s[i] ^= U8TO64_LE(in);
	
	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= c_keccak_round_constants[i];
	}
}

__device__ void
keccak_hash_init(keccak_hash_state *S) {
	memset(S, 0, sizeof(*S));
}

__device__ void
keccak_hash_update(keccak_hash_state *S, const uint8_t *in, size_t inlen) {
	size_t want;

	/* handle the previous data */
	if (S->leftover) {
		want = (KECCAK_HASH_BLOCK_SIZE - S->leftover);
		want = (want < inlen) ? want : inlen;
		memcpy(S->buffer + S->leftover, in, want);
		S->leftover += (uint32_t)want;
		if (S->leftover < KECCAK_HASH_BLOCK_SIZE)
			return;
		in += want;
		inlen -= want;
		keccak_block(S, S->buffer);
	}

	/* handle the current data */
	while (inlen >= KECCAK_HASH_BLOCK_SIZE) {
		keccak_block(S, in);
		in += KECCAK_HASH_BLOCK_SIZE;
		inlen -= KECCAK_HASH_BLOCK_SIZE;
	}

	/* handle leftover data */
	S->leftover = (uint32_t)inlen;
	if (S->leftover)
		memcpy(S->buffer, in, S->leftover);
}

__device__ void
keccak_hash_finish(keccak_hash_state *S, uint8_t *hash) {
	size_t i;

	S->buffer[S->leftover] = 0x01;
	memset(S->buffer + (S->leftover + 1), 0, KECCAK_HASH_BLOCK_SIZE - (S->leftover + 1));
	S->buffer[KECCAK_HASH_BLOCK_SIZE - 1] |= 0x80;
	keccak_block(S, S->buffer);

	for (i = 0; i < KECCAK_HASH_DIGEST_SIZE; i += 8) {
		U64TO8_LE(&hash[i], S->state[i / 8]);
	}
}

// ---------------------------- END keccak functions ------------------------------------

// ---------------------------- BEGIN PBKDF2 functions ------------------------------------

typedef struct scrypt_hmac_state_t {
	keccak_hash_state inner, outer;
} scrypt_hmac_state;


__device__ void
scrypt_hash(keccak_hash_digest hash, const uint8_t *m, size_t mlen) {
	keccak_hash_state st;
	keccak_hash_init(&st);
	keccak_hash_update(&st, m, mlen);
	keccak_hash_finish(&st, hash);
}

/* hmac */
__device__ void
scrypt_hmac_init(scrypt_hmac_state *st, const uint8_t *key, size_t keylen) {
	uint8_t pad[KECCAK_HASH_BLOCK_SIZE] = {0};
	size_t i;

	keccak_hash_init(&st->inner);
	keccak_hash_init(&st->outer);

	if (keylen <= KECCAK_HASH_BLOCK_SIZE) {
		/* use the key directly if it's <= blocksize bytes */
		memcpy(pad, key, keylen);
	} else {
		/* if it's > blocksize bytes, hash it */
		scrypt_hash(pad, key, keylen);
	}

	/* inner = (key ^ 0x36) */
	/* h(inner || ...) */
	for (i = 0; i < KECCAK_HASH_BLOCK_SIZE; i++)
		pad[i] ^= 0x36;
	keccak_hash_update(&st->inner, pad, KECCAK_HASH_BLOCK_SIZE);

	/* outer = (key ^ 0x5c) */
	/* h(outer || ...) */
	for (i = 0; i < KECCAK_HASH_BLOCK_SIZE; i++)
		pad[i] ^= (0x5c ^ 0x36);
	keccak_hash_update(&st->outer, pad, KECCAK_HASH_BLOCK_SIZE);
}

__device__ void
scrypt_hmac_update(scrypt_hmac_state *st, const uint8_t *m, size_t mlen) {
	/* h(inner || m...) */
	keccak_hash_update(&st->inner, m, mlen);
}

__device__ void
scrypt_hmac_finish(scrypt_hmac_state *st, keccak_hash_digest mac) {
	/* h(inner || m) */
	keccak_hash_digest innerhash;
	keccak_hash_finish(&st->inner, innerhash);

	/* h(outer || h(inner || m)) */
	keccak_hash_update(&st->outer, innerhash, sizeof(innerhash));
	keccak_hash_finish(&st->outer, mac);
}

/*
 * Special version where N = 1
 *  - mikaelh
 */
__device__ void
scrypt_pbkdf2_1(const uint8_t *password, size_t password_len, const uint8_t *salt, size_t salt_len, uint8_t *out, size_t bytes) {
	scrypt_hmac_state hmac_pw, hmac_pw_salt, work;
	keccak_hash_digest ti, u;
	uint8_t be[4];
	uint32_t i, /*j,*/ blocks;
//	uint64_t c;
	
	/* bytes must be <= (0xffffffff - (SCRYPT_HASH_DIGEST_SIZE - 1)), which they will always be under scrypt */

	/* hmac(password, ...) */
	scrypt_hmac_init(&hmac_pw, password, password_len);

	/* hmac(password, salt...) */
	hmac_pw_salt = hmac_pw;
	scrypt_hmac_update(&hmac_pw_salt, salt, salt_len);

	blocks = ((uint32_t)bytes + (KECCAK_HASH_DIGEST_SIZE - 1)) / KECCAK_HASH_DIGEST_SIZE;
	for (i = 1; i <= blocks; i++) {
		/* U1 = hmac(password, salt || be(i)) */
		U32TO8_BE(be, i);
		work = hmac_pw_salt;
		scrypt_hmac_update(&work, be, 4);
		scrypt_hmac_finish(&work, ti);
		memcpy(u, ti, sizeof(u));

		memcpy(out, ti, (bytes > KECCAK_HASH_DIGEST_SIZE) ? KECCAK_HASH_DIGEST_SIZE : bytes);
		out += KECCAK_HASH_DIGEST_SIZE;
		bytes -= KECCAK_HASH_DIGEST_SIZE;
	}
}

// ---------------------------- END PBKDF2 functions ------------------------------------

static __device__ uint32_t cuda_swab32(uint32_t x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__global__ void cuda_pre_keccak512(uint32_t *g_idata, uint32_t nonce)
{
    nonce        +=       (blockIdx.x * blockDim.x) + threadIdx.x; 
    g_idata      += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);

    uint32_t data[20];

#pragma unroll 19
    for (int i=0; i <19; ++i)
        data[i] = cuda_swab32(pdata[i]);
    data[19] = cuda_swab32(nonce);

    scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)data, 80, (uint8_t*)g_idata, 128);
}


__global__ void cuda_post_keccak512(uint32_t *g_odata, uint32_t *g_hash, uint32_t nonce)
{
    nonce        +=       (blockIdx.x * blockDim.x) + threadIdx.x; 
    g_odata      += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);
    g_hash       +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);

    uint32_t data[20];

#pragma unroll 19
    for (int i=0; i <19; ++i)
        data[i] = cuda_swab32(pdata[i]);
    data[19] = cuda_swab32(nonce);

    scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)g_odata, 128, (uint8_t*)g_hash, 32);
}

//
// callable host code to initialize constants and to call kernels
//

extern "C" void prepare_keccak512(int thr_id, uint32_t host_pdata[20])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        hipMemcpyToSymbol(HIP_SYMBOL(c_keccak_round_constants), host_keccak_round_constants, sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice);
        init[thr_id] = true;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

extern "C" void pre_keccak512(int thr_id, int stream, uint32_t nonce, int throughput)
{
    dim3 block(128);
    dim3 grid((throughput+127)/128);

    cuda_pre_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_idata[stream][thr_id], nonce);
}

extern "C" void post_keccak512(int thr_id, int stream, uint32_t nonce, uint32_t hash[8], int throughput)
{
    dim3 block(128);
    dim3 grid((throughput+127)/128);

    cuda_post_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_odata[stream][thr_id], context_hash[stream][thr_id], nonce);

    unsigned int mem_size = throughput * sizeof(uint32_t) * 8;

    // copy device memory to host
    checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                    hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}
