#include "hip/hip_runtime.h"
//
//  =============== BLAKE part on nVidia GPU ======================
//
// This is the generic "default" implementation when no architecture
// specific implementation is available in the kernel.
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//
// TODO: CUDA porting work remains to be done.
//

#include <map>
#include <stdint.h>

#include "salsa_kernel.h"
#include "miner.h"

#if 1
#include "sph_blake.h"
#else
typedef uint32_t sph_u32;
#endif

__constant__ uint64_t ptarget64[4];
__constant__ uint32_t pdata[20];

// define some error checking macros
#undef checkCudaErrors

#if WIN32
#define DELIMITER '/'
#else
#define DELIMITER '/'
#endif
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) calling '%s' (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), #x, __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

// #define SPH_SMALL_FOOTPRINT_BLAKE 1

#if SPH_SMALL_FOOTPRINT && !defined SPH_SMALL_FOOTPRINT_BLAKE
#define SPH_SMALL_FOOTPRINT_BLAKE   1
#endif

#if SPH_SMALL_FOOTPRINT_BLAKE
#define SPH_COMPACT_BLAKE_32   1
#endif

#ifdef _MSC_VER
#pragma warning (disable: 4146)
#endif

static __device__ sph_u32 cuda_sph_bswap32(sph_u32 x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

/**
 * Encode a 32-bit value into the provided buffer (big endian convention).
 *
 * @param dst   the destination buffer
 * @param val   the 32-bit value to encode
 */
static __device__ void
cuda_sph_enc32be(void *dst, sph_u32 val)
{
	*(sph_u32 *)dst = cuda_sph_bswap32(val);
}

/**
 * Decode a 32-bit value from the provided buffer (big endian convention).
 *
 * @param src   the source buffer (32-bit aligned)
 * @return  the decoded value
 */
static __device__ sph_u32
cuda_sph_dec32be(const void *src)
{
	return cuda_sph_bswap32(*(const sph_u32 *)src);
}

__constant__ sph_u32 IV256[8];

const sph_u32 host_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

#if SPH_COMPACT_BLAKE_32

__constant__ unsigned sigma[14][16];

const unsigned host_sigma[14][16] = {
	{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
	{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 },
	{  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 },
	{  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 },
	{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 },
	{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 },
	{  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 },
	{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13,  0 },
	{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
	{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 }
};

#endif

#define Z00   0
#define Z01   1
#define Z02   2
#define Z03   3
#define Z04   4
#define Z05   5
#define Z06   6
#define Z07   7
#define Z08   8
#define Z09   9
#define Z0A   A
#define Z0B   B
#define Z0C   C
#define Z0D   D
#define Z0E   E
#define Z0F   F

#define Z10   E
#define Z11   A
#define Z12   4
#define Z13   8
#define Z14   9
#define Z15   F
#define Z16   D
#define Z17   6
#define Z18   1
#define Z19   C
#define Z1A   0
#define Z1B   2
#define Z1C   B
#define Z1D   7
#define Z1E   5
#define Z1F   3

#define Z20   B
#define Z21   8
#define Z22   C
#define Z23   0
#define Z24   5
#define Z25   2
#define Z26   F
#define Z27   D
#define Z28   A
#define Z29   E
#define Z2A   3
#define Z2B   6
#define Z2C   7
#define Z2D   1
#define Z2E   9
#define Z2F   4

#define Z30   7
#define Z31   9
#define Z32   3
#define Z33   1
#define Z34   D
#define Z35   C
#define Z36   B
#define Z37   E
#define Z38   2
#define Z39   6
#define Z3A   5
#define Z3B   A
#define Z3C   4
#define Z3D   0
#define Z3E   F
#define Z3F   8

#define Z40   9
#define Z41   0
#define Z42   5
#define Z43   7
#define Z44   2
#define Z45   4
#define Z46   A
#define Z47   F
#define Z48   E
#define Z49   1
#define Z4A   B
#define Z4B   C
#define Z4C   6
#define Z4D   8
#define Z4E   3
#define Z4F   D

#define Z50   2
#define Z51   C
#define Z52   6
#define Z53   A
#define Z54   0
#define Z55   B
#define Z56   8
#define Z57   3
#define Z58   4
#define Z59   D
#define Z5A   7
#define Z5B   5
#define Z5C   F
#define Z5D   E
#define Z5E   1
#define Z5F   9

#define Z60   C
#define Z61   5
#define Z62   1
#define Z63   F
#define Z64   E
#define Z65   D
#define Z66   4
#define Z67   A
#define Z68   0
#define Z69   7
#define Z6A   6
#define Z6B   3
#define Z6C   9
#define Z6D   2
#define Z6E   8
#define Z6F   B

#define Z70   D
#define Z71   B
#define Z72   7
#define Z73   E
#define Z74   C
#define Z75   1
#define Z76   3
#define Z77   9
#define Z78   5
#define Z79   0
#define Z7A   F
#define Z7B   4
#define Z7C   8
#define Z7D   6
#define Z7E   2
#define Z7F   A

#define Z80   6
#define Z81   F
#define Z82   E
#define Z83   9
#define Z84   B
#define Z85   3
#define Z86   0
#define Z87   8
#define Z88   C
#define Z89   2
#define Z8A   D
#define Z8B   7
#define Z8C   1
#define Z8D   4
#define Z8E   A
#define Z8F   5

#define Z90   A
#define Z91   2
#define Z92   8
#define Z93   4
#define Z94   7
#define Z95   6
#define Z96   1
#define Z97   5
#define Z98   F
#define Z99   B
#define Z9A   9
#define Z9B   E
#define Z9C   3
#define Z9D   C
#define Z9E   D
#define Z9F   0

#define Mx(r, i)    Mx_(Z ## r ## i)
#define Mx_(n)      Mx__(n)
#define Mx__(n)     M ## n

#define CSx(r, i)   CSx_(Z ## r ## i)
#define CSx_(n)     CSx__(n)
#define CSx__(n)    CS ## n

#define CS0   SPH_C32(0x243F6A88)
#define CS1   SPH_C32(0x85A308D3)
#define CS2   SPH_C32(0x13198A2E)
#define CS3   SPH_C32(0x03707344)
#define CS4   SPH_C32(0xA4093822)
#define CS5   SPH_C32(0x299F31D0)
#define CS6   SPH_C32(0x082EFA98)
#define CS7   SPH_C32(0xEC4E6C89)
#define CS8   SPH_C32(0x452821E6)
#define CS9   SPH_C32(0x38D01377)
#define CSA   SPH_C32(0xBE5466CF)
#define CSB   SPH_C32(0x34E90C6C)
#define CSC   SPH_C32(0xC0AC29B7)
#define CSD   SPH_C32(0xC97C50DD)
#define CSE   SPH_C32(0x3F84D5B5)
#define CSF   SPH_C32(0xB5470917)

#if SPH_COMPACT_BLAKE_32

__constant__ sph_u32 CS[16];

const sph_u32 host_CS[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#endif

#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#if SPH_COMPACT_BLAKE_32

#define ROUND_S(r)   do { \
		GS(M[sigma[r][0x0]], M[sigma[r][0x1]], \
			CS[sigma[r][0x0]], CS[sigma[r][0x1]], V0, V4, V8, VC); \
		GS(M[sigma[r][0x2]], M[sigma[r][0x3]], \
			CS[sigma[r][0x2]], CS[sigma[r][0x3]], V1, V5, V9, VD); \
		GS(M[sigma[r][0x4]], M[sigma[r][0x5]], \
			CS[sigma[r][0x4]], CS[sigma[r][0x5]], V2, V6, VA, VE); \
		GS(M[sigma[r][0x6]], M[sigma[r][0x7]], \
			CS[sigma[r][0x6]], CS[sigma[r][0x7]], V3, V7, VB, VF); \
		GS(M[sigma[r][0x8]], M[sigma[r][0x9]], \
			CS[sigma[r][0x8]], CS[sigma[r][0x9]], V0, V5, VA, VF); \
		GS(M[sigma[r][0xA]], M[sigma[r][0xB]], \
			CS[sigma[r][0xA]], CS[sigma[r][0xB]], V1, V6, VB, VC); \
		GS(M[sigma[r][0xC]], M[sigma[r][0xD]], \
			CS[sigma[r][0xC]], CS[sigma[r][0xD]], V2, V7, V8, VD); \
		GS(M[sigma[r][0xE]], M[sigma[r][0xF]], \
			CS[sigma[r][0xE]], CS[sigma[r][0xF]], V3, V4, V9, VE); \
	} while (0)

#else

#define ROUND_S(r)   do { \
		GS(Mx(r, 0), Mx(r, 1), CSx(r, 0), CSx(r, 1), V0, V4, V8, VC); \
		GS(Mx(r, 2), Mx(r, 3), CSx(r, 2), CSx(r, 3), V1, V5, V9, VD); \
		GS(Mx(r, 4), Mx(r, 5), CSx(r, 4), CSx(r, 5), V2, V6, VA, VE); \
		GS(Mx(r, 6), Mx(r, 7), CSx(r, 6), CSx(r, 7), V3, V7, VB, VF); \
		GS(Mx(r, 8), Mx(r, 9), CSx(r, 8), CSx(r, 9), V0, V5, VA, VF); \
		GS(Mx(r, A), Mx(r, B), CSx(r, A), CSx(r, B), V1, V6, VB, VC); \
		GS(Mx(r, C), Mx(r, D), CSx(r, C), CSx(r, D), V2, V7, V8, VD); \
		GS(Mx(r, E), Mx(r, F), CSx(r, E), CSx(r, F), V3, V4, V9, VE); \
	} while (0)

#endif

#define DECL_STATE32 \
	sph_u32 H0, H1, H2, H3, H4, H5, H6, H7; \
	sph_u32 S0, S1, S2, S3, T0, T1;

#define READ_STATE32(state)   do { \
		H0 = (state)->H[0]; \
		H1 = (state)->H[1]; \
		H2 = (state)->H[2]; \
		H3 = (state)->H[3]; \
		H4 = (state)->H[4]; \
		H5 = (state)->H[5]; \
		H6 = (state)->H[6]; \
		H7 = (state)->H[7]; \
		S0 = (state)->S[0]; \
		S1 = (state)->S[1]; \
		S2 = (state)->S[2]; \
		S3 = (state)->S[3]; \
		T0 = (state)->T0; \
		T1 = (state)->T1; \
	} while (0)

#define WRITE_STATE32(state)   do { \
		(state)->H[0] = H0; \
		(state)->H[1] = H1; \
		(state)->H[2] = H2; \
		(state)->H[3] = H3; \
		(state)->H[4] = H4; \
		(state)->H[5] = H5; \
		(state)->H[6] = H6; \
		(state)->H[7] = H7; \
		(state)->S[0] = S0; \
		(state)->S[1] = S1; \
		(state)->S[2] = S2; \
		(state)->S[3] = S3; \
		(state)->T0 = T0; \
		(state)->T1 = T1; \
	} while (0)

#if SPH_COMPACT_BLAKE_32

#define COMPRESS32   do { \
		sph_u32 M[16]; \
		sph_u32 V0, V1, V2, V3, V4, V5, V6, V7; \
		sph_u32 V8, V9, VA, VB, VC, VD, VE, VF; \
		unsigned r; \
		V0 = H0; \
		V1 = H1; \
		V2 = H2; \
		V3 = H3; \
		V4 = H4; \
		V5 = H5; \
		V6 = H6; \
		V7 = H7; \
		V8 = S0 ^ CS0; \
		V9 = S1 ^ CS1; \
		VA = S2 ^ CS2; \
		VB = S3 ^ CS3; \
		VC = T0 ^ CS4; \
		VD = T0 ^ CS5; \
		VE = T1 ^ CS6; \
		VF = T1 ^ CS7; \
		M[0x0] = cuda_sph_dec32be(buf +  0); \
		M[0x1] = cuda_sph_dec32be(buf +  4); \
		M[0x2] = cuda_sph_dec32be(buf +  8); \
		M[0x3] = cuda_sph_dec32be(buf + 12); \
		M[0x4] = cuda_sph_dec32be(buf + 16); \
		M[0x5] = cuda_sph_dec32be(buf + 20); \
		M[0x6] = cuda_sph_dec32be(buf + 24); \
		M[0x7] = cuda_sph_dec32be(buf + 28); \
		M[0x8] = cuda_sph_dec32be(buf + 32); \
		M[0x9] = cuda_sph_dec32be(buf + 36); \
		M[0xA] = cuda_sph_dec32be(buf + 40); \
		M[0xB] = cuda_sph_dec32be(buf + 44); \
		M[0xC] = cuda_sph_dec32be(buf + 48); \
		M[0xD] = cuda_sph_dec32be(buf + 52); \
		M[0xE] = cuda_sph_dec32be(buf + 56); \
		M[0xF] = cuda_sph_dec32be(buf + 60); \
		for (r = 0; r < 8; r ++) \
			ROUND_S(r); \
		H0 ^= S0 ^ V0 ^ V8; \
		H1 ^= S1 ^ V1 ^ V9; \
		H2 ^= S2 ^ V2 ^ VA; \
		H3 ^= S3 ^ V3 ^ VB; \
		H4 ^= S0 ^ V4 ^ VC; \
		H5 ^= S1 ^ V5 ^ VD; \
		H6 ^= S2 ^ V6 ^ VE; \
		H7 ^= S3 ^ V7 ^ VF; \
	} while (0)

#else

#define COMPRESS32   do { \
		sph_u32 M0, M1, M2, M3, M4, M5, M6, M7; \
		sph_u32 M8, M9, MA, MB, MC, MD, ME, MF; \
		sph_u32 V0, V1, V2, V3, V4, V5, V6, V7; \
		sph_u32 V8, V9, VA, VB, VC, VD, VE, VF; \
		V0 = H0; \
		V1 = H1; \
		V2 = H2; \
		V3 = H3; \
		V4 = H4; \
		V5 = H5; \
		V6 = H6; \
		V7 = H7; \
		V8 = S0 ^ CS0; \
		V9 = S1 ^ CS1; \
		VA = S2 ^ CS2; \
		VB = S3 ^ CS3; \
		VC = T0 ^ CS4; \
		VD = T0 ^ CS5; \
		VE = T1 ^ CS6; \
		VF = T1 ^ CS7; \
		M0 = cuda_sph_dec32be(buf +  0); \
		M1 = cuda_sph_dec32be(buf +  4); \
		M2 = cuda_sph_dec32be(buf +  8); \
		M3 = cuda_sph_dec32be(buf + 12); \
		M4 = cuda_sph_dec32be(buf + 16); \
		M5 = cuda_sph_dec32be(buf + 20); \
		M6 = cuda_sph_dec32be(buf + 24); \
		M7 = cuda_sph_dec32be(buf + 28); \
		M8 = cuda_sph_dec32be(buf + 32); \
		M9 = cuda_sph_dec32be(buf + 36); \
		MA = cuda_sph_dec32be(buf + 40); \
		MB = cuda_sph_dec32be(buf + 44); \
		MC = cuda_sph_dec32be(buf + 48); \
		MD = cuda_sph_dec32be(buf + 52); \
		ME = cuda_sph_dec32be(buf + 56); \
		MF = cuda_sph_dec32be(buf + 60); \
		ROUND_S(0); \
		ROUND_S(1); \
		ROUND_S(2); \
		ROUND_S(3); \
		ROUND_S(4); \
		ROUND_S(5); \
		ROUND_S(6); \
		ROUND_S(7); \
		H0 ^= S0 ^ V0 ^ V8; \
		H1 ^= S1 ^ V1 ^ V9; \
		H2 ^= S2 ^ V2 ^ VA; \
		H3 ^= S3 ^ V3 ^ VB; \
		H4 ^= S0 ^ V4 ^ VC; \
		H5 ^= S1 ^ V5 ^ VD; \
		H6 ^= S2 ^ V6 ^ VE; \
		H7 ^= S3 ^ V7 ^ VF; \
	} while (0)

#endif

__constant__ sph_u32 salt_zero_small[4];

const sph_u32 host_salt_zero_small[4] = { 0, 0, 0, 0 };

static __device__ void
cuda_blake32_init(sph_blake_small_context *sc,
	const sph_u32 *iv, const sph_u32 *salt)
{
	memcpy(sc->H, iv, 8 * sizeof(sph_u32));
	memcpy(sc->S, salt, 4 * sizeof(sph_u32));
	sc->T0 = sc->T1 = 0;
	sc->ptr = 0;
}

static __device__ void
cuda_blake32(sph_blake_small_context *sc, const void *data, size_t len)
{
	unsigned char *buf;
	size_t ptr;
	DECL_STATE32

	buf = sc->buf;
	ptr = sc->ptr;
	if (len < (sizeof sc->buf) - ptr) {
		memcpy(buf + ptr, data, len);
		ptr += len;
		sc->ptr = ptr;
		return;
	}

	READ_STATE32(sc);
	while (len > 0) {
		size_t clen;

		clen = (sizeof sc->buf) - ptr;
		if (clen > len)
			clen = len;
		memcpy(buf + ptr, data, clen);
		ptr += clen;
		data = (const unsigned char *)data + clen;
		len -= clen;
		if (ptr == sizeof sc->buf) {
			if ((T0 = SPH_T32(T0 + 512)) < 512)
				T1 = SPH_T32(T1 + 1);
			COMPRESS32;
			ptr = 0;
		}
	}
	WRITE_STATE32(sc);
	sc->ptr = ptr;
}

static __device__ void
cuda_blake32_close(sph_blake_small_context *sc,
	unsigned ub, unsigned n, void *dst, size_t out_size_w32)
{
	union {
		unsigned char buf[64];
		sph_u32 dummy;
	} u;
	size_t ptr, k;
	unsigned bit_len;
	unsigned z;
	sph_u32 th, tl;
	unsigned char *out;

	ptr = sc->ptr;
	bit_len = ((unsigned)ptr << 3) + n;
	z = 0x80 >> n;
	u.buf[ptr] = ((ub & -z) | z) & 0xFF;
	tl = sc->T0 + bit_len;
	th = sc->T1;
	if (ptr == 0 && n == 0) {
		sc->T0 = SPH_C32(0xFFFFFE00);
		sc->T1 = SPH_C32(0xFFFFFFFF);
	} else if (sc->T0 == 0) {
		sc->T0 = SPH_C32(0xFFFFFE00) + bit_len;
		sc->T1 = SPH_T32(sc->T1 - 1);
	} else {
		sc->T0 -= 512 - bit_len;
	}
	if (bit_len <= 446) {
		memset(u.buf + ptr + 1, 0, 55 - ptr);
		if (out_size_w32 == 8)
			u.buf[55] |= 1;
		cuda_sph_enc32be(u.buf + 56, th);
		cuda_sph_enc32be(u.buf + 60, tl);
		cuda_blake32(sc, u.buf + ptr, 64 - ptr);
	} else {
		memset(u.buf + ptr + 1, 0, 63 - ptr);
		cuda_blake32(sc, u.buf + ptr, 64 - ptr);
		sc->T0 = SPH_C32(0xFFFFFE00);
		sc->T1 = SPH_C32(0xFFFFFFFF);
		memset(u.buf, 0, 56);
		if (out_size_w32 == 8)
			u.buf[55] = 1;
		cuda_sph_enc32be(u.buf + 56, th);
		cuda_sph_enc32be(u.buf + 60, tl);
		cuda_blake32(sc, u.buf, 64);
	}
	out = (unsigned char*)dst;
	for (k = 0; k < out_size_w32; k ++)
		cuda_sph_enc32be(out + (k << 2), sc->H[k]);
}


static __device__ uint32_t cuda_swab32(uint32_t x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__global__ void cuda_blake256_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
    uint32_t data[20];
    uint64_t out[4];

#pragma unroll 19
    for (int i=0; i < 19; ++i) data[i] = pdata[i];
    data[19] = cuda_swab32(nonce + ((blockIdx.x * blockDim.x) + threadIdx.x));

    sph_blake_small_context ctx;
    cuda_blake32_init(&ctx, IV256, salt_zero_small);
    cuda_blake32(&ctx, data, 80);
    cuda_blake32_close(&ctx, 0, 0, out, 8);

    if (validate)
    {
        g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
#pragma unroll 4
        for (int i=0; i < 4; ++i) g_out[i] = out[i];
    }

    uint64_t *g_good64 = (uint64_t*)g_good;
    if (out[3] <=  ptarget64[3]) {
        if (out[3] < g_good64[3]) {
            g_good64[3] = out[3];
            g_good64[2] = out[2];
            g_good64[1] = out[1];
            g_good64[0] = out[0];
            g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
        }
    }
}

static std::map<int, uint32_t *> context_good[2];

extern "C" void default_prepare_blake256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(IV256), host_IV256, sizeof(host_IV256), 0, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(salt_zero_small), host_salt_zero_small, sizeof(host_salt_zero_small), 0, hipMemcpyHostToDevice));
#if SPH_COMPACT_BLAKE_32
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CS), host_CS, sizeof(host_CS), 0, hipMemcpyHostToDevice));
#endif
	
	// allocate pinned host memory for good hashes
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

        init[thr_id] = true;
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

extern "C" bool default_do_blake256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    bool success = true;
  
    checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

    cuda_blake256_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

    // copy hashes from device memory to host (ALL hashes, lots of data...)
    if (do_d2h && hash != NULL) {
        size_t mem_size = throughput * sizeof(uint32_t) * 8;
        checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }
    else if (hash != NULL) {
        // asynchronous copy of winning nonce (just 4 bytes...)
        checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }

        // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}
